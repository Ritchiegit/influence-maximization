
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

extern "C"
{
#define AUXILIARY_NODE_ID -1

    typedef struct node
    {
        int id;
        node *prev;
        node *next;
        __device__ node(int id) : id(id){};
    } node_t;

    __global__ void init_rng(int nthreads, hiprandState *states, unsigned long long seed, unsigned long long offset)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if (id >= nthreads)
            return;
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(seed, id, offset, &states[id]);
    }

    __global__ void generate_rr_sets(float *data, int *rows, int *cols, bool *out, int *nodeHistogram, int numNodes, int numNonZeros, int numSets, hiprandState *states)
    {
        const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
        if (tid < numSets)
        {
            hiprandState *state = &states[tid];

            /* Because C does not give us the luxury of dynamic arrays, to imitate the
        behavior of a stack, I am using a linked list*/
            int randomNodeId = ceil(numNodes * hiprand_uniform(state)) - 1;
            node *stack = new node(randomNodeId);
            node *auxiliary = new node(AUXILIARY_NODE_ID);
            auxiliary->next = stack;
            stack->prev = auxiliary;

            // Returns false when stack is NULL
            while (stack->id != AUXILIARY_NODE_ID)
            {
                // pop from stack
                int currentNodeId = stack->id;
                node *temp = stack;
                stack = stack->prev;
                free(temp);

                // If current is not in visited
                if (!out[tid * numNodes + currentNodeId])
                {
                    out[tid * numNodes + currentNodeId] = true;
                    atomicAdd(&nodeHistogram[currentNodeId], 1);

                    int dataStart = rows[currentNodeId];
                    int dataEnd = rows[currentNodeId + 1];

                    for (unsigned int i = dataStart; i < dataEnd; i++)
                    {
                        if (hiprand_uniform(state) < data[i])
                        {
                            // append to stack
                            stack->next = new node(cols[i]);
                            stack->next->prev = stack;
                            stack = stack->next;
                        }
                    }
                }
            }
            free(auxiliary);
        }
    }

    __global__ void count_node_to_node_intersections(int *counts, bool *batch, int num_rows, int num_nodes)
    {
        int row = blockDim.x * blockIdx.x + threadIdx.x;
        int node_y = blockDim.y * blockIdx.y + threadIdx.y;
        int node_z = blockDim.z * blockIdx.z + threadIdx.z;

        if (row < num_rows && node_y < num_nodes && node_z < num_nodes)
        {
            if (batch[row * num_nodes + node_y] && batch[row * num_nodes + node_z])
            {
                atomicAdd(&counts[node_y * num_nodes + node_z], 1);
            }
        }
    }
}
