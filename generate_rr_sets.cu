
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

typedef struct node
{
    int id;
    node *prev;
    node *next;
    __device__ node(int id) : id(id){};
} node_t;

__global__ void init_rng(int nthreads, hiprandState *states, unsigned long long seed, unsigned long long offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= nthreads)
        return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, id, offset, &states[id]);
}

__global__ void generate_rr_sets(float *data, int *rows, int *cols, int *out, int numNodes, int numNonZeros, int theta, hiprandState *states)
{
    const unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < theta)
    {
        hiprandState state = states[tid];

        /* Because C does not give us the luxury of dynamic arrays, to imitate the
        behavior of a stack, I am using a linked list*/
        int randomNodeId = ceil(numNodes * hiprand_uniform(&state)) - 1;
        node *stack = new node(randomNodeId);

        // Returns false when stack is NULL
        while (stack)
        {
            // pop from stack
            int currentNodeId = stack->id;
            node *temp = stack;
            stack = stack->prev;
            free(temp);

            // If current is not in visited
            if (!out[tid * numNodes + currentNodeId])
            {
                out[tid * numNodes + currentNodeId] = 1; // visited.add(currentNodeId)

                int dataStart = rows[currentNodeId];
                int dataEnd = rows[currentNodeId + 1];

                for (unsigned int i = dataStart; i < dataEnd; i++)
                {
                    if (hiprand_uniform(&state) < data[i])
                    {
                        // append to stack
                        stack->next = new node(cols[i]);
                    }
                }
            }
        }
    }
}
